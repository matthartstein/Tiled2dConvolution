#include "hip/hip_runtime.h"
/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/
 
__constant__ float M_c[FILTER_SIZE][FILTER_SIZE];

__global__ void convolution(Matrix N, Matrix P)
{
	/********************************************************************
	Determine input and output indexes of each thread
	Load a tile of the input image to shared memory
	Apply the filter on the input image tile
	Write the compute values to the output image at the correct indexes
	********************************************************************/

    //INSERT KERNEL CODE HERE
    
    __shared__ float N_c[BLOCK_SIZE][BLOCK_SIZE];
    
	int row;
	int col;
	int sRow = threadIdx.y + (FILTER_SIZE / 2);
	int sCol = threadIdx.x + (FILTER_SIZE / 2);
	int gRow = blockIdx.y * TILE_SIZE + threadIdx.y;
	int gCol = blockIdx.x * TILE_SIZE + threadIdx.x;
	float pVal = 0.0;
	int i = 0;
	int j = 0;

	if (gRow >= 0 && gRow < N.height && gCol >= 0 && gCol < N.width)
		N_c[sRow][sCol] = N.elements[gRow * N.width + gCol];
	else
		N_c[sRow][sCol] = 0.0;

	if (threadIdx.y < (FILTER_SIZE / 2) && threadIdx.x < (FILTER_SIZE / 2)) {
		row = (blockIdx.y + 1) * TILE_SIZE + threadIdx.y;
		col = (blockIdx.x + 1) * TILE_SIZE + threadIdx.x;

		if (row < N.height && col < N.width && row >= 0 && col >= 0)
			N_c[sRow + TILE_SIZE][sCol + TILE_SIZE] = N.elements[row * N.width + col];
		else
			N_c[sRow + TILE_SIZE][sCol + TILE_SIZE] = 0.0;
	}

	if (threadIdx.y >= TILE_SIZE - (FILTER_SIZE / 2) && threadIdx.x < (FILTER_SIZE / 2)) {
		row = (blockIdx.y - 1) * TILE_SIZE + threadIdx.y;
		col = (blockIdx.x + 1) * TILE_SIZE + threadIdx.x; 

		if (row < N.height && col < N.width && row >= 0 && col >= 0)
			N_c[sRow - TILE_SIZE][sCol + TILE_SIZE] = N.elements[row * N.width + col];
		else
			N_c[sRow - TILE_SIZE][sCol + TILE_SIZE] = 0.0;
	}

	if (threadIdx.y < (FILTER_SIZE / 2) && threadIdx.x >= TILE_SIZE - (FILTER_SIZE / 2)) {
		row = (blockIdx.y + 1) * TILE_SIZE + threadIdx.y;
		col = (blockIdx.x - 1) * TILE_SIZE + threadIdx.x;

		if (row < N.height && col < N.width && row >= 0 && col >= 0)
			N_c[sRow + TILE_SIZE][sCol - TILE_SIZE] = N.elements[row * N.width + col];
		else
			N_c[sRow + TILE_SIZE][sCol - TILE_SIZE] = 0.0;
	}

	if (threadIdx.y >= TILE_SIZE - (FILTER_SIZE / 2) && threadIdx.x >= TILE_SIZE - (FILTER_SIZE / 2)) {
		row = (blockIdx.y - 1) * TILE_SIZE + threadIdx.y;
		col = (blockIdx.x - 1) * TILE_SIZE + threadIdx.x;

		if (row < N.height && col < N.width && row >= 0 && col >= 0)
			N_c[sRow - TILE_SIZE][sCol - TILE_SIZE] = N.elements[row * N.width + col];
		else
			N_c[sRow - TILE_SIZE][sCol - TILE_SIZE] = 0.0;
	}

	if (threadIdx.y < (FILTER_SIZE / 2)) {
		row = (blockIdx.y + 1) * TILE_SIZE + threadIdx.y;
		col = (blockIdx.x) * TILE_SIZE + threadIdx.x;

		if (row < N.height && col < N.width && row >= 0 && col >= 0)
			N_c[sRow + TILE_SIZE][sCol - 0] = N.elements[row * N.width + col];
		else
			N_c[sRow + TILE_SIZE][sCol - 0] = 0.0;
	}

	if (threadIdx.x < (FILTER_SIZE / 2)) {
		row = (blockIdx.y) * TILE_SIZE + threadIdx.y;
		col = (blockIdx.x + 1) * TILE_SIZE + threadIdx.x;

		if (row < N.height && col < N.width && row >= 0 && col >= 0)
			N_c[sRow][sCol + TILE_SIZE] = N.elements[row * N.width + col];
		else
			N_c[sRow][sCol + TILE_SIZE] = 0.0;
	}

	if (threadIdx.y >= TILE_SIZE - (FILTER_SIZE / 2)) {
		row = (blockIdx.y - 1) * TILE_SIZE + threadIdx.y;
		col = (blockIdx.x) * TILE_SIZE + threadIdx.x;

		if (row < N.height && col < N.width && row >= 0 && col >= 0)
			N_c[sRow - TILE_SIZE][sCol] = N.elements[row * N.width + col];
		else
			N_c[sRow - TILE_SIZE][sCol] = 0.0;
	}

	if (threadIdx.x >= TILE_SIZE - (FILTER_SIZE / 2)) {
		row = (blockIdx.y) * TILE_SIZE + threadIdx.y;
		col = (blockIdx.x - 1) * TILE_SIZE + threadIdx.x;
		
		if (row < N.height && col < N.width && row >= 0 && col >= 0)
			N_c[sRow - 0][sCol - TILE_SIZE] = N.elements[row * N.width + col];
		else
			N_c[sRow - 0][sCol - TILE_SIZE] = 0.0;
	}

	__syncthreads();

	if (gRow < P.height && gCol < P.width) {
		for (int z = -2; z < 3; z++) {
			for (int x = -2; x < 3; x++) {
				i = z + 2;
				j = x + 2;
				pVal += N_c[sRow + z][sCol + x] * M_c[i][j];
			}
		}
		P.elements[gRow * P.width + gCol] = pVal;
	}
}